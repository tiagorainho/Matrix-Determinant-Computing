#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>

// (row, col, order)
#define idx(x,y,order)(x*order+y)

#define KRED  "\x1B[31m"
#define KGRN  "\x1B[32m"
#define KNRM  "\x1B[0m"

inline void switch_col(double *matrix, int col1, int col2, int order) {
    double aux;
    
    for(int i = 0; i < order; i++) 
    {
        aux = matrix[idx(i,col1,order)];
        matrix[idx(i,col1,order)] = matrix[idx(i,col2,order)];
        matrix[idx(i,col2,order)] = aux;
    }
}

void determinantOnHostRows(double *matrices, int numberOfMatrix, double *determinant, int order)
{
    int sign = 1;
    double ratio = 1;
    
    for(int n = 0; n < numberOfMatrix; n++)
    {
        double *matrix = &matrices[n * order * order];
        determinant[n] = 1;

        //for each col
        for(int i = 0; i < order; i++) 
        {
            // check if the col can be used, otherwise, switch that col
            if(matrix[idx(i,i,order)] == 0) {
                bool determinantIsZero = true;
                for(int j = i+1; j < order; j++) 
                {
                    if(matrix[idx(i,j,order)] != 0) 
                    {
                        switch_col(matrix, i, j, order);
                        sign = (sign == 1) ? -1: 1;
                        determinantIsZero = false;
                        break;
                    }
                }
                if(determinantIsZero)
                {
                    determinant[n] = 0;
                    break;
                }                
            }
            //for each col
            for(int j = i + 1; j < order; j++) {
                ratio = matrix[idx(i,j,order)] / matrix[idx(i,i,order)];
                for(int k = 0; k < order; k++) {
                    matrix[idx(k,j,order)] = matrix[idx(k,j,order)] - ratio * matrix[idx(k,i,order)];
                }
            }
            determinant[n] *= matrix[idx(i,i,order)];
        }
        determinant[n] *= sign;
    }
}

__global__ void determinantOnGPURows(double *mat, double *determinant, int order)
{
    extern __shared__ double tmp[];  

    unsigned int rowNumber = threadIdx.x + threadIdx.y * blockDim.x;
    unsigned int matrixNumber = blockIdx.x + blockIdx.y * gridDim.x;
    unsigned int size = order * order;

    double *matrix = &mat[matrixNumber * size];

    int sign = 1;
    determinant[matrixNumber] = 1;
    
    //for each col
    for(int i = 0; i < order; i++)
    {
        //swap col if necessary
        if(matrix[idx(i,i,order)] == 0) {
            bool determinantIsZero = true;
            for(int j= i + 1; j < order; j++) {
                if(matrix[idx(i,j,order)] != 0) {
                    double t = matrix[idx(rowNumber,i,order)];
                    matrix[idx(rowNumber,i,order)] = matrix[idx(rowNumber,j,order)];
                    matrix[idx(rowNumber,j,order)] = t;
                    sign = (sign == 1) ? -1 : 1;
                    determinantIsZero = false;
                    break;
                }
            } 
            //determinant is 0
            if(determinantIsZero)
            {            
                if(rowNumber == 0)
                    determinant[matrixNumber] = 0;
                return;               
            }
        }
        __syncthreads();

        //Read all necessary values
        tmp[rowNumber] = matrix[idx(i,rowNumber,order)];
        __syncthreads();

        //For all other cols
        for(int j = i + 1; j < order; j++)
        {
            double ratio = tmp[j] / matrix[idx(i,i,order)];

            //process corresponding col
            matrix[idx(rowNumber,j,order)] = matrix[idx(rowNumber,j,order)]-ratio*matrix[idx(rowNumber,i,order)];
        }
        //__syncthreads();
    }

    //calculate determinant
    if(rowNumber == 0)
    {
        determinant[matrixNumber] = sign;
        for(int i = 0; i < order; i++)
            determinant[matrixNumber] = determinant[matrixNumber] * matrix[idx(i,i,order)];
    }
}

void checkResult(double *cpuRef, double *gpuRef, int nDeterminants)
{
   
    bool match = 1;
    for(int i = 0; i < nDeterminants; i++)
    {
        double epsilon = (1 - cpuRef[i] / gpuRef[i]) * 100;
        if(epsilon < 0)
            epsilon = -epsilon;

        if (epsilon > 0.000001)
        {
            match = 0;
            printf("%sError: Matrix %3d - host %.8e \t gpu %.8e\n%s", KRED, i + 1, cpuRef[i], gpuRef[i], KNRM);
            break;
        }

        //printf("%sCorrect: Matrix %3d - host %.3e \t gpu %.3e\n%s", KGRN, i + 1, cpuRef[i], gpuRef[i], KNRM);
    }

    if (match)
        printf("Determinants match.\n\n");
    else
        printf("Determinants do not match.\n\n");
}

int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);

    // process cli 
    int opt;
    char * fileName;

    do {
        switch((opt = getopt(argc, argv, "f:h"))) {
            case 'f':
                fileName = optarg;
                break;
                
            case 'h':
                printf("-f      --- filename\n");
                break;
        }
    }
    while(opt != -1);

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up data size of matrix
    int order;
    int numberOfMatrix;
    
    FILE * ptrFile = fopen(fileName, "r"); 
    if(ptrFile == NULL) {
        fprintf(stderr,"Error opening file");
        printf("%s\n", fileName);
        exit(EXIT_FAILURE);
    }     
    size_t size = fread(&numberOfMatrix, sizeof(unsigned int), 1, ptrFile);
    if(size != 1)
    {
        fprintf(stderr,"Error reading the number of matrix in the file\n");
        exit(EXIT_FAILURE);
    }

    size = fread(&order, sizeof(unsigned int), 1, ptrFile);
    if(size != 1)
    {
        fprintf(stderr,"Error reading order from file\n");
        exit(EXIT_FAILURE);
    }

    printf("Filename: %s\nNumber of matrices: %d\nMatrices order: %d\n", fileName, numberOfMatrix, order);

    //host memory
    double *h_matrices = (double *)malloc(order*order*numberOfMatrix*sizeof(double));
    size = fread(h_matrices, sizeof(double), order*order*numberOfMatrix, ptrFile);
    if(size != order*order*numberOfMatrix)
    {
        fprintf(stderr,"Error matrices from file\n");
        exit(EXIT_FAILURE);
    }
    

    double determinantRefCPU[numberOfMatrix];
    double determinantRefGPU[numberOfMatrix];

    int nBytesMatrices = order * order * numberOfMatrix * sizeof(double);
    int nBytesDeterminants = numberOfMatrix * sizeof(double);


    // malloc device global memory
    double *d_matrices;
    double *d_determinant;

    CHECK(hipMalloc((void **)&d_matrices, nBytesMatrices));
    CHECK(hipMalloc((void **)&d_determinant, nBytesDeterminants));

    // transfer data from host to device
    CHECK(hipMemcpy(d_matrices, h_matrices, nBytesMatrices, hipMemcpyHostToDevice));

    dim3 block(order, 1);
    dim3 grid(numberOfMatrix, 1);

    double iStart;
    double iElaps;
    

    // calculate determinant at host side
    iStart = seconds();
    determinantOnHostRows(h_matrices, numberOfMatrix, determinantRefCPU, order);
    iElaps = seconds() - iStart;
    printf("determinantOnHostRows elapsed %f sec\n", iElaps);

    // invoke kernel at host side
    iStart = seconds();
    determinantOnGPURows<<<grid, block, order * sizeof(double)>>>(d_matrices, d_determinant, order);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("determinantOnGPURows <<<(%d,%d), (%d,%d)>>> elapsed %f sec\n", grid.x,
           grid.y,
           block.x, block.y, iElaps);

    // check kernel error
    CHECK(hipGetLastError());

    // copy kernel result back to host side
    CHECK(hipMemcpy(determinantRefGPU, d_determinant, nBytesDeterminants, hipMemcpyDeviceToHost));

    // check device results
    checkResult(determinantRefCPU, determinantRefGPU, numberOfMatrix);

    //clear previous results
    memset(determinantRefCPU, 0, nBytesDeterminants);
    memset(determinantRefGPU, 0, nBytesDeterminants);
    CHECK(hipMemset(d_determinant, 0, nBytesDeterminants));


    // free device global memory
    CHECK(hipFree(d_matrices));
    CHECK(hipFree(d_determinant));

    // reset device
    CHECK(hipDeviceReset());

    return (0);
}
